#include "Image.h"


 int* Image::GetGpuRPnt()
 {
	 return ret;
 }
 int* Image::GetGpuCPnt()
 {
	 return cort;
}



void Image::SetDataGpuR(int *d){
 if(ret!=NULL)
 hipFree(ret);
 hipMalloc((void**)&ret, W*H*sizeof(int));
 hipMemcpy(ret, d, W*H*sizeof(int), hipMemcpyHostToDevice);
}

void Image::SetDataGpuC(int R, int S){
 if(cort!=NULL)
 hipFree(cort);
 hipMalloc((void**)&cort, R*S*sizeof(int));
}


